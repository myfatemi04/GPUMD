#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan, Ville Vierimaa, Mikko Ervasti, and Ari Harju
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/

#include "fitness.cuh"
#include "parameters.cuh"
#include "snes.cuh"
#include "utilities/error.cuh"
#include "utilities/main_common.cuh"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

void print_welcome_information(void);

int main(int argc, char* argv[])
{
  print_welcome_information();
  print_gpu_information();
  // int number_of_inputs = 1;
  int number_of_inputs = get_number_of_input_directories();

  for (int n = 0; n < number_of_inputs; ++n) {
    // char *input_directory = "/home/gsk6me/out/PbTe_NEP3_baseline";
    char input_directory[200];
    int count = scanf("%s", input_directory);
    PRINT_SCANF_ERROR(count, 1, "reading error for input directory.");

    print_line_1();
    printf("Run simulation for '%s'.\n", input_directory);
    print_line_2();

    clock_t time_begin = clock();
    Parameters para(input_directory);
    Fitness fitness(input_directory, para);
    clock_t time_finish = clock();

    float time_used = (time_finish - time_begin) / float(CLOCKS_PER_SEC);
    print_line_1();
    printf("Time used for initialization = %f s.\n", time_used);
    print_line_2();

    time_begin = clock();
    SNES snes(input_directory, para, &fitness);
    time_finish = clock();

    time_used = (time_finish - time_begin) / float(CLOCKS_PER_SEC);
    print_line_1();
    printf("Time used for training = %f s.\n", time_used);
    print_line_2();
  }

  print_line_1();
  printf("Finished running nep.\n");
  print_line_2();

  return EXIT_SUCCESS;
}

void print_welcome_information(void)
{
  printf("\n");
  printf("***************************************************************\n");
  printf("*                 Welcome to use GPUMD                        *\n");
  printf("*    (Graphics Processing Units Molecular Dynamics)           *\n");
  printf("*                    Version 3.3.1                            *\n");
  printf("*              This is the nep executable                     *\n");
  printf("***************************************************************\n");
  printf("\n");
}
